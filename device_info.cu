/**
 * CUDA Device Information Utility
 * 
 * This program displays detailed information about available CUDA devices.
 * Useful for debugging CUDA installation and device compatibility.
 * 
 * Compile with: nvcc -o device_info device_info.cu
 * Run with: ./device_info
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

/**
 * Print detailed information about a CUDA device
 * 
 * @param deviceId - The device ID to query
 */
void printDeviceInfo(int deviceId) {
    hipDeviceProp_t prop;
    hipError_t error = hipGetDeviceProperties(&prop, deviceId);
    
    if (error != hipSuccess) {
        printf("Error getting device properties for device %d: %s\n", 
               deviceId, hipGetErrorString(error));
        return;
    }
    
    // Calculate CUDA cores based on compute capability
    int cudaCores = 0;
    if (prop.major == 7) {
        if (prop.minor == 5) {
            cudaCores = prop.multiProcessorCount * 128; // Turing architecture
        } else if (prop.minor == 0) {
            cudaCores = prop.multiProcessorCount * 64;  // Volta architecture
        }
    } else if (prop.major == 8) {
        cudaCores = prop.multiProcessorCount * 128; // Ampere architecture
    } else if (prop.major == 9) {
        cudaCores = prop.multiProcessorCount * 144; // Hopper architecture
    }
    
    printf("\n=== Device %d: %s ===\n", deviceId, prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("CUDA Cores: %d\n", cudaCores);
    printf("Global Memory: %lu MB\n", prop.totalGlobalMem / (1024 * 1024));
    printf("Shared Memory per Block: %lu KB\n", prop.sharedMemPerBlock / 1024);
    printf("Registers per Block: %d\n", prop.regsPerBlock);
    printf("Warp Size: %d\n", prop.warpSize);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Threads per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Max Blocks per Multiprocessor: %d\n", prop.maxBlocksPerMultiProcessor);
    printf("Number of Multiprocessors: %d\n", prop.multiProcessorCount);
    printf("Max Grid Size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Max Block Size: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Memory Clock Rate: %d MHz\n", prop.memoryClockRate / 1000);
    printf("Memory Bus Width: %d bits\n", prop.memoryBusWidth);
    printf("ECC Enabled: %s\n", prop.ECCEnabled ? "Yes" : "No");
    printf("Unified Memory: %s\n", prop.unifiedAddressing ? "Yes" : "No");
    printf("Concurrent Kernels: %s\n", prop.concurrentKernels ? "Yes" : "No");
    printf("Compute Mode: %d\n", prop.computeMode);
}

int main() {
    printf("CUDA Device Information Utility\n");
    printf("===============================\n");
    
    // Get number of devices
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(error));
        return 1;
    }
    
    if (deviceCount == 0) {
        printf("No CUDA-capable devices found.\n");
        return 1;
    }
    
    printf("Found %d CUDA device(s):\n", deviceCount);
    
    // Print information for each device
    for (int i = 0; i < deviceCount; i++) {
        printDeviceInfo(i);
    }
    
    // Get current device
    int currentDevice;
    error = hipGetDevice(&currentDevice);
    if (error == hipSuccess) {
        printf("\nCurrent device: %d\n", currentDevice);
    }
    
    printf("\nCUDA Runtime Version: %d.%d\n", CUDART_VERSION/1000, (CUDART_VERSION%1000)/10);
    
    return 0;
} 